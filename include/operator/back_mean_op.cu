#include "hip/hip_runtime.h"
#include "caffe2/operators/back_mean_op.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

__global__ void BackMeanKernel(const int N, const float ratio, const float* Xdata, float* Ydata, bool* maskdata) {
  const float scale = 1. / (1. - ratio);
  CUDA_1D_KERNEL_LOOP(i, N) {
    maskdata[i] = (Ydata[i] > ratio);
    Ydata[i] = Xdata[i] * scale * maskdata[i];
  }
}

}  // namespace

template <>
bool BackMeanOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  auto* mask = Output(1);
  Y->Resize(X.dims());
  mask->Resize(X.dims());
  if (is_test_) {
    if (Y != &X) {
      context_.Copy<float, HIPContext, HIPContext>(
          X.size(), X.data<float>(), Y->mutable_data<float>());
    }
    return true;
  } else {
    // We do a simple trick here: since hiprand cannot generate random
    // boolean numbers, we will generate into dY and write the result to
    // mask.
    float* Ydata = Y->mutable_data<float>();
    CAFFE_ENFORCE(X.data<float>() != Ydata, "In-place GPU BackMean is broken");
    CURAND_ENFORCE(
        hiprandGenerateUniform(context_.curand_generator(), Ydata, X.size()));
    BackMeanKernel<<<CAFFE_GET_BLOCKS(X.size()), CAFFE_CUDA_NUM_THREADS,
                    0, context_.hip_stream()>>>(
        X.size(), ratio_, X.data<float>(), Ydata, mask->mutable_data<bool>());
    return true;
  }
}

namespace {

__global__ void BackMeanGradientKernel(const int N, const float* dYdata, const bool* maskdata, const float scale, float* dXdata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dXdata[i] = dYdata[i] * maskdata[i] * scale;
  }
}

}  // namespace

template <>
bool BackMeanGradientOp<float, HIPContext>::RunOnDevice() {
  auto& dY = Input(0);
  auto& mask = Input(1);
  auto* dX = Output(0);
  DCHECK_EQ(dY.size(), mask.size());
  dX->Resize(dY.dims());
  if (is_test_) {
    if (dX != &dY) {
      context_.Copy<float, HIPContext, HIPContext>(
          dY.size(), dY.data<float>(), dX->mutable_data<float>());
    }
    return true;
  } else {
    const float scale = 1. / (1. - ratio_);
    BackMeanGradientKernel<<<CAFFE_GET_BLOCKS(dY.size()),
                            CAFFE_CUDA_NUM_THREADS,
                            0, context_.hip_stream()>>>(
        dY.size(), dY.data<float>(), mask.data<bool>(), scale,
        dX->mutable_data<float>());
    return true;
  }
}


namespace {

REGISTER_HIP_OPERATOR(BackMean, BackMeanOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(BackMeanGrad, BackMeanGradientOp<float, HIPContext>);

}  // namespace

}  // namespace caffe2
