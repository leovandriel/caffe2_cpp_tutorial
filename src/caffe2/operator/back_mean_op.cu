#include "hip/hip_runtime.h"
#include "caffe2/operator/back_mean_op.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

int back_mean_strip(std::vector<TIndex> &dims, int count) {
  auto size = 1;
  while (count--) {
    size *= dims.back();
    dims.pop_back();
  }
  return size;
}

namespace {

__global__ void ZeroKernel(const int N, float* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
   Y[i] = 0;
  }
}

__global__ void BackMeanKernel(const int N, const int C, const float D, const float* X, float* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i / C] += X[i] / D;
  }
}

}  // namespace

template <>
bool BackMeanOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);
  auto dims = X.dims();
  auto size = back_mean_strip(dims, count_);
  Y->Resize(dims);
  if (Y->size() > 0) {
    ZeroKernel<<<CAFFE_GET_BLOCKS(Y->size()), CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
      Y->size(), Y->mutable_data<float>());
    BackMeanKernel<<<CAFFE_GET_BLOCKS(X.size()), CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
      X.size(), size, (float)size, X.data<float>(), Y->mutable_data<float>());
  }
  return true;
}

namespace {

__global__ void BackMeanGradientKernel(const int N, const int C, const float D, const float* dY, float* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = dY[i / C] / D;
  }
}

}  // namespace

template <>
bool BackMeanGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& dY = Input(1);
  auto* dX = Output(0);
  dX->ResizeLike(X);
  auto dims = X.dims();
  auto size = back_mean_strip(dims, count_);
  DCHECK_EQ(dY.size() * size, dX->size());
  if (dY.size() > 0) {
    BackMeanGradientKernel<<<CAFFE_GET_BLOCKS(dX->size()), CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
      dX->size(), size, (float)size, dY.data<float>(), dX->mutable_data<float>());
  }
  return true;
}


namespace {

REGISTER_HIP_OPERATOR(BackMean, BackMeanOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(BackMeanGradient, BackMeanGradientOp<float, HIPContext>);

}  // namespace

}  // namespace caffe2
